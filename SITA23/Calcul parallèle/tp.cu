#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel pour le produit de convolution 1D
__global__ void conv1D(int n, float *x, int p, float *h, float *y) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calcul de l'index global
    if (idx < n) {
        float sum = 0.0;
        for (int k = max(0, idx - p + 1); k <= idx; k++) {
            sum += x[k] * h[idx - k];
        }
        y[idx] = sum; // Stocke le résultat dans le vecteur de sortie
    }
}

// Kernel pour le produit de convolution 2D
__global__ void conv2D(int m, int n, float *x, int p, int q, float *h, float *y) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Index ligne
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Index colonne

    if (i < m && j < n) {
        float sum = 0.0;
        for (int k = max(0, i - p + 1); k <= i; k++) {
            for (int l = max(0, j - q + 1); l <= j; l++) {
                sum += x[k * n + l] * h[(i - k) * q + (j - l)];
            }
        }
        y[i * n + j] = sum; // Stocke le résultat dans la matrice de sortie
    }
}

int main() {
    // Dimensions du problème pour l'exemple
    const int n = 10; // Taille du vecteur 1D
    const int m = 4, nc = 4; // Dimensions de la matrice 2D
    const int p = 3, q = 3; // Dimensions des filtres

    // Allocation mémoire sur l'hôte
    float h_x1D[n] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10}; // Vecteur d'entrée
    float h_h1D[p] = {1, 0, -1}; // Filtre 1D
    float h_y1D[n] = {0}; // Résultat 1D

    float h_x2D[m * nc] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16}; // Matrice d'entrée
    float h_h2D[p * q] = {0, 1, 0, 1, -4, 1, 0, 1, 0}; // Filtre 2D
    float h_y2D[m * nc] = {0}; // Résultat 2D

    // Allocation mémoire sur le device
    float *d_x1D, *d_h1D, *d_y1D;
    float *d_x2D, *d_h2D, *d_y2D;

    hipMalloc(&d_x1D, n * sizeof(float));
    hipMalloc(&d_h1D, p * sizeof(float));
    hipMalloc(&d_y1D, n * sizeof(float));

    hipMalloc(&d_x2D, m * nc * sizeof(float));
    hipMalloc(&d_h2D, p * q * sizeof(float));
    hipMalloc(&d_y2D, m * nc * sizeof(float));

    // Copie des données sur le device
    hipMemcpy(d_x1D, h_x1D, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_h1D, h_h1D, p * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_x2D, h_x2D, m * nc * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_h2D, h_h2D, p * q * sizeof(float), hipMemcpyHostToDevice);

    // Lancement des kernels
    dim3 block1D(256);
    dim3 grid1D((n + block1D.x - 1) / block1D.x);

    conv1D<<<grid1D, block1D>>>(n, d_x1D, p, d_h1D, d_y1D);

    dim3 block2D(16, 16);
    dim3 grid2D((nc + block2D.x - 1) / block2D.x, (m + block2D.y - 1) / block2D.y);

    conv2D<<<grid2D, block2D>>>(m, nc, d_x2D, p, q, d_h2D, d_y2D);

    // Copie des résultats vers l'hôte
    hipMemcpy(h_y1D, d_y1D, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_y2D, d_y2D, m * nc * sizeof(float), hipMemcpyDeviceToHost);

    // Affichage des résultats
    printf("Résultat de la convolution 1D :\n");
    for (int i = 0; i < n; i++) {
        printf("%.2f ", h_y1D[i]);
    }
    printf("\n\n");

    printf("Résultat de la convolution 2D :\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < nc; j++) {
            printf("%.2f ", h_y2D[i * nc + j]);
        }
        printf("\n");
    }

    // Libération de la mémoire
    hipFree(d_x1D);
    hipFree(d_h1D);
    hipFree(d_y1D);

    hipFree(d_x2D);
    hipFree(d_h2D);
    hipFree(d_y2D);

    return 0;
}
